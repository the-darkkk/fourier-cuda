#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include "fourier-lib.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

__global__ void calculateCoefficientsKernel(double*, double*, const double*, const double*, int, int, double);
__global__ void calculateSeriesKernel(double*, const double*, const double*, const double*, int, int, double);

FourierCudaCalculator::FourierCudaCalculator() {
	selectedDeviceIndex = -1; // initializing the selected device index
};

FourierCudaCalculator::~FourierCudaCalculator() {};

std::vector<std::string> FourierCudaCalculator::GetAvailableDevices() {
	std::vector<std::string> devices;
	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	
	if (error_id != hipSuccess) {
		return devices;
	}
    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        devices.push_back(std::string(props.name));
    }
    
	return devices;
};

bool FourierCudaCalculator::SelectDevice(int deviceId) {
    hipError_t error_id = hipSetDevice(deviceId);

    if (error_id == hipSuccess) {
        this->selectedDeviceIndex = deviceId;
        return true;
    }
    else {
        this->selectedDeviceIndex = -1;
        return false;
    }
};
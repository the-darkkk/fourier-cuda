#include "hip/hip_runtime.h"
#include <cmath>
#include "fourier-lib.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

__global__ void calculateCoefficientsKernel(double*, double*, const double*, const double*, int, int, double);
__global__ void calculateSeriesKernel(double*, const double*, const double*, const double*, int, int, double);
__global__ void normalizeCoefficientsKernel(double*, double*, const double*, const double*, int, int);

FourierCudaCalculator::FourierCudaCalculator() {
	selectedDeviceIndex = -1; // initializing the selected device index
};

FourierCudaCalculator::~FourierCudaCalculator() {};

std::vector<std::string> FourierCudaCalculator::GetAvailableDevices() {
	std::vector<std::string> devices;
	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	
	if (error_id != hipSuccess) {
		return devices;
	}
    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        devices.push_back(std::string(props.name));
    }
    
	return devices;
};

bool FourierCudaCalculator::SelectDevice(int deviceId) {
    hipError_t error_id = hipSetDevice(deviceId);

    if (error_id == hipSuccess) {
        this->selectedDeviceIndex = deviceId;
        return true;
    }
    else {
        this->selectedDeviceIndex = -1;
        return false;
    }
};

Result FourierCudaCalculator::Calculate(const Params& params, const std::vector<double>& x_values, const std::vector<double>& y_values) {
    Result result;
    const double PI = 3.14159265358979323846;   // a kludge bcs M_PI somewhy does not work here
    if (selectedDeviceIndex < 0) {
        result.isSuccess = false;
        result.errorMessage = "GPU device is not selected.";
        return result;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    try {
        const int Ne = x_values.size();
        const int Ng = params.numHarmonics;
        const double Tp = x_values.back() - x_values.front();   // assuming that period is the given range of digits
        const double w = 2.0 * PI / Tp;     // calculating the edge frequency

        double a0_sum = 0.0; // calculate a0
        for (const auto& y : y_values) a0_sum += y;
        const double a0 = a0_sum / Ne;

        // allocate the vram for calculation
        double* d_x, * d_y, * d_G, * d_D, * d_a, * d_b, * d_Yg;
        const size_t size_Ne = Ne * sizeof(double);     // allocate the size needed for the input dots array
        const size_t size_Ng = (Ng + 1) * sizeof(double);   // allocate the size needed for the output harmonics array
        hipMalloc(&d_x, size_Ne);
        hipMalloc(&d_y, size_Ne);
        hipMalloc(&d_G, size_Ng); 
        hipMalloc(&d_D, size_Ng);
        hipMalloc(&d_a, size_Ng); 
        hipMalloc(&d_b, size_Ng);
        hipMalloc(&d_Yg, size_Ne); 

        hipEventRecord(start); // start time measurement - needed for time telemetry

        // move the data from ram to vram
        hipMemcpy(d_x, x_values.data(), size_Ne, hipMemcpyHostToDevice);
        hipMemcpy(d_y, y_values.data(), size_Ne, hipMemcpyHostToDevice);
        hipMemset(d_G, 0, size_Ng); // zeroing the memory - i guess it's needed
        hipMemset(d_D, 0, size_Ng);

        const int threadsPerBlock = 256; // defining the constant

        calculateCoefficientsKernel << <Ng, threadsPerBlock >> > (d_G, d_D, d_x, d_y, Ne, Ng, w);   // launching a parallel fourier coefficients calculation

        const int blocksForNorm = (Ng + threadsPerBlock - 1) / threadsPerBlock;
        normalizeCoefficientsKernel << <blocksForNorm, threadsPerBlock >> > (d_a, d_b, d_G, d_D, Ne, Ng);

        hipMemcpy(&d_a[0], &a0, sizeof(double), hipMemcpyHostToDevice);   // copy the results back to cpu
        
        const int blocksPerGrid = (Ne + threadsPerBlock - 1) / threadsPerBlock;     // modifying the blocks count
        calculateSeriesKernel << <blocksPerGrid, threadsPerBlock >> > (d_Yg, d_x, d_a, d_b, Ne, Ng, w);     // launching a parallel calculation to reconctruct func

        hipEventRecord(stop); // stop the time measurement
        hipEventSynchronize(stop);
        hipEventElapsedTime(&result.executionTimeMs, start, stop);

        // retrieve the results from gpu
        result.calculatedY.resize(Ne);
        result.a_coeffs.resize(Ng + 1);
        result.b_coeffs.resize(Ng + 1);
        hipMemcpy(result.calculatedY.data(), d_Yg, size_Ne, hipMemcpyDeviceToHost);
        std::vector<double> G_host(Ng + 1), D_host(Ng + 1);
        hipMemcpy(G_host.data(), d_G, size_Ng, hipMemcpyDeviceToHost);
        hipMemcpy(D_host.data(), d_D, size_Ng, hipMemcpyDeviceToHost);

        // free the vram
        hipFree(d_x); hipFree(d_y); hipFree(d_G); hipFree(d_D);
        hipFree(d_a); hipFree(d_b); hipFree(d_Yg);

        // final calculations
        result.a_coeffs[0] = a0;
        result.b_coeffs[0] = 0; // b0 always zero
        result.c_amplitudes.resize(Ng + 1);
        result.c_amplitudes[0] = 0;
        for (int k = 1; k <= Ng; ++k) {
            result.a_coeffs[k] = G_host[k] * 2.0 / Ne;
            result.b_coeffs[k] = D_host[k] * 2.0 / Ne;
            result.c_amplitudes[k] = std::sqrt(result.a_coeffs[k] * result.a_coeffs[k] + result.b_coeffs[k] * result.b_coeffs[k]);
        }
        result.isSuccess = true;

    }
    catch (const std::exception& e) {
        result.isSuccess = false;
        result.errorMessage = e.what();
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return result;
};